#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <cstdlib>
#include <iostream>
#include <functional>
#include <cmath>
using namespace std;
#define SIZE 100

/*
 * @struct varianceshifteop
 * @brief a unary function that shifts input data
 * by their mean and computes the squares of them
 */
struct varianceshifteop : unary_function<float, float>
{
    varianceshifteop(float m) : mean(m)
    { /* no-op */ }

    const float mean;
    
    __device__ float operator()(float data) const
    {
        return pow(data - mean, 2.0f);
    }
};

int main()
{
  // generate some random data on the host
  thrust::host_vector<int> h_vec(SIZE);
  for(int i=0; i<h_vec.size(); i++)
    h_vec[i]=rand()%RAND_MAX;
  // transfer to device
  thrust::device_vector<int> d_vec = h_vec;
   // sum on device
  float mean = thrust::reduce(d_vec.begin(), d_vec.end(), 0.0f, thrust::plus<int>()) / d_vec.size();
  int max_val = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::maximum<int>());
  int min_val = thrust::reduce(d_vec.begin(), d_vec.end(), RAND_MAX, thrust::minimum<int>());
  float variance = thrust::transform_reduce(d_vec.begin(), d_vec.end(), varianceshifteop(mean), 0.0f, thrust::plus<int>()) / (d_vec.size() - 1);
  // standard dev is just a sqrt away
  float stdv = sqrtf(variance);
  cout<<"Mean = "<<mean<<"  Max = "<<max_val<<"  Min = "<<min_val<<" Standard Deviation = "<<stdv<<endl;
  return 0;
}